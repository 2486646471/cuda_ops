#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREAD_PER_BLOCK 256
#define WARP_SIZE 32

template<int num_per_block>
__global__ void reduce(float* d_input, float* d_output) {
    float sum = 0.f;
    int tid = threadIdx.x;
    float* d_input_start = d_input + blockIdx.x * num_per_block;

    for (int i = 0; i < num_per_block / THREAD_PER_BLOCK; i++) {
        sum += d_input_start[tid + i * THREAD_PER_BLOCK];
    }
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;

    for (int s = 16; s > 0; s >>= 1) {
        if (lane_id < s) {
            sum += __shfl_down_sync(0xffffffff, sum, s);
        }
    }

    __shared__ float warpSums[32];
    if (lane_id == 0) {
        warpSums[warp_id] = sum;
    } 
    __syncthreads();

    if (warp_id == 0) {
        sum = (lane_id < blockDim.x / 32) ? warpSums[lane_id] : 0.f;
        for (int s = 16; s > 0; s >>= 1) {
            if (lane_id < s) {
                sum += __shfl_down_sync(0xffffffff, sum, s);
            }
        }
    }
    // 写入结果
    if (tid == 0) {
        d_output[blockIdx.x] = warpSums[0];
    }
}

// 检查结果
bool check(float *out, float *res, int n) {
    const float epsilon = 0.01;
    for (int i = 0; i < n; i++) {
        if (fabs(out[i] - res[i]) > epsilon) return false;
    }
    return true;
}

void randomMatrix(float* matrix, int N) {
    for (int i = 0; i < N; i++) {
        matrix[i] = 2.0 * (float)drand48() - 1.0;
    }
}

template<int num_per_block, int block_num>
void cpuReduce(float* h_input, float* res) {
    for (int i = 0; i < block_num; i++) {
        float sum = 0;
        for (int j = 0; j < num_per_block; j++) {
            sum += h_input[i * num_per_block + j];
        }
        res[i] = sum;
    }
}

int main() {
    const int N = 32 * 1024 * 1024;
    const int block_num = 1024;
    const int num_per_block = N / block_num;  // 32768

    // 分配主机内存
    float *h_input = (float*)malloc(N * sizeof(float));
    float *h_output = (float*)malloc(block_num * sizeof(float));

    // 初始化输入数据
    randomMatrix(h_input, N);

    // 分配设备内存
    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, block_num * sizeof(float));

    // 数据拷贝到设备
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // 启动核函数
    dim3 grid(block_num);
    dim3 block(THREAD_PER_BLOCK);
    reduce<num_per_block><<<grid, block>>>(d_input, d_output);

    // 拷贝结果回主机
    hipMemcpy(h_output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_output[i]);
    }
    printf("\n");
    // 计算正确结果
    float *res = (float*)malloc(block_num * sizeof(float));
    cpuReduce<num_per_block, block_num>(h_input, res);
    for (int i = 0; i < 10; i++) {
        printf("%f ", res[i]);
    }
    printf("\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_output[i]);
    }
    // 验证结果
    if (check(h_output, res, block_num)) {
        printf("结果正确！\n");
    } else {
        printf("结果错误！\n");
    }

    // 释放内存
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    free(res);

    return 0;
}